#include "hip/hip_runtime.h"
#include "cuReduceSum_implement.h"

template<class T>
struct SharedMemory
{
    __device__ inline operator       T*()
    {
        extern __shared__ int __smem[];
        return (T*)__smem;
    }
};

template <class T, unsigned int blockSize, bool nIsPow2>
__global__ void reduceFindSum_kernel(T *g_idata, T *g_odata, uint n)
{
    T *sdata = SharedMemory<T>();

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockSize*2 + threadIdx.x;
    unsigned int gridSize = blockSize*2*gridDim.x;
    
    T mySum = 0.f;

    // we reduce multiple elements per thread.  The number is determined by the 
    // number of active thread blocks (via gridDim).  More blocks will result
    // in a larger gridSize and therefore fewer elements per thread
    while (i < n)
    {         
        mySum += g_idata[i];
        // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
        if (nIsPow2 || i + blockSize < n) 
            mySum += g_idata[i+blockSize];  
        i += gridSize;
    } 

    // each thread puts its local sum into shared memory 
    sdata[tid] = mySum;
    __syncthreads();


    // do reduction in shared mem
    if (blockSize >= 512 && tid < 256) { 
        mySum += sdata[tid + 256];
		sdata[tid] = mySum; 
	}
	__syncthreads(); 
    
    if (blockSize >= 256 && tid < 128) { 
		mySum += sdata[tid + 128]; 
		sdata[tid] = mySum; 
	} 
	__syncthreads(); 
		
    if (blockSize >= 128 && tid <  64) { 
		mySum += sdata[tid +  64]; 
		sdata[tid] = mySum; 
	}
	__syncthreads(); 
        
        // fully unroll reduction within a single warp
	if ((blockSize >=  64) && (tid < 32)) {
		mySum += sdata[tid + 32];
		sdata[tid] = mySum;
	}
	__syncthreads(); 
        
	if ((blockSize >=  32) && (tid < 16)) { 
	   mySum += sdata[tid + 16];
		sdata[tid] = mySum;
	}
	__syncthreads(); 
	
	if ((blockSize >=  16) && (tid <  8)) {
		mySum += sdata[tid +  8];
		sdata[tid] = mySum;
	}
	__syncthreads(); 
        
	if ((blockSize >=   8) && (tid <  4)) { 
		mySum += sdata[tid +  4];
		sdata[tid] = mySum;
	}
	__syncthreads(); 
        
	if ((blockSize >=   4) && (tid <  2)) { 
		mySum += sdata[tid +  2];
		sdata[tid] = mySum;
	}
	__syncthreads(); 
	
	if ((blockSize >=   2) && ( tid <  1)) {
		mySum += sdata[tid +  1];
		sdata[tid] = mySum;
	}
	__syncthreads();         
   
    // write result for this block to global mem 
    if (tid == 0) 
        g_odata[blockIdx.x] = sdata[0];
}
    
template <class T>
void cuReduceFindSum(T *dst, T *src, 
    uint n, uint nBlocks, uint nThreads)
{
	dim3 dimBlock(nThreads, 1, 1);
    dim3 dimGrid(nBlocks, 1, 1);
	int smemSize = (nThreads <= 32) ? 2 * nThreads * sizeof(T) : nThreads * sizeof(T);
	
	if (isPow2(n)) {
		switch (nThreads)
		{
		case 512:
			reduceFindSum_kernel<T, 512, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case 256:
			reduceFindSum_kernel<T, 256, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case 128:
			reduceFindSum_kernel<T, 128, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case 64:
			reduceFindSum_kernel<T, 64, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case 32:
			reduceFindSum_kernel<T, 32, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case 16:
			reduceFindSum_kernel<T, 16, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case  8:
			reduceFindSum_kernel<T,  8, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case  4:
			reduceFindSum_kernel<T,  4, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case  2:
			reduceFindSum_kernel<T,  2, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case  1:
			reduceFindSum_kernel<T,  1, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		}
	}
	else {
		switch (nThreads)
		{
		case 512:
			reduceFindSum_kernel<T, 512, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case 256:
			reduceFindSum_kernel<T, 256, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case 128:
			reduceFindSum_kernel<T, 128, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case 64:
			reduceFindSum_kernel<T, 64, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case 32:
			reduceFindSum_kernel<T, 32, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case 16:
			reduceFindSum_kernel<T, 16, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case  8:
			reduceFindSum_kernel<T,  8, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case  4:
			reduceFindSum_kernel<T,  4, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case  2:
			reduceFindSum_kernel<T,  2, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		case  1:
			reduceFindSum_kernel<T,  1, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, n); break;
		}
	}
}

template
void cuReduceFindSum<int>(int *dst, int *src, 
    uint n, uint nBlocks, uint nThreads);
	
template
void cuReduceFindSum<float>(float *dst, float *src, 
    uint n, uint nBlocks, uint nThreads);
