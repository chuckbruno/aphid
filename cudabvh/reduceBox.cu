#include "hip/hip_runtime.h"
#include "reduceBox_implement.h"
#include <bvh_math.cu>

inline __device__ void copyVola(volatile Aabb * dst, const Aabb & src)
{
    dst->low.x = src.low.x;
    dst->low.y = src.low.y;
    dst->low.z = src.low.z;
    dst->high.x = src.high.x;
    dst->high.y = src.high.y;
    dst->high.z = src.high.z;
}

template <unsigned int blockSize, bool nIsPow2>
__global__ void reduceAabbByAabb_kernel(Aabb *g_idata, Aabb *g_odata, unsigned int n)
{
    extern __shared__ Aabb sdata[];

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockSize*2 + threadIdx.x;
    unsigned int gridSize = blockSize*2*gridDim.x;
    
    Aabb mySum; resetAabb(mySum);

    // we reduce multiple elements per thread.  The number is determined by the 
    // number of active thread blocks (via gridDim).  More blocks will result
    // in a larger gridSize and therefore fewer elements per thread
    while (i < n)
    {         
        expandAabb(mySum, g_idata[i]);
        // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
        if (nIsPow2 || i + blockSize < n) 
            expandAabb(mySum, g_idata[i+blockSize]);  
        i += gridSize;
    } 

    // each thread puts its local sum into shared memory 
    sdata[tid] = mySum;
    __syncthreads();


    // do reduction in shared mem
    if (blockSize >= 512) { 
        if (tid < 256) { 
            expandAabb(mySum, sdata[tid + 256]);
            sdata[tid] = mySum; 
        } 
        __syncthreads(); 
    }
    if (blockSize >= 256) { 
        if (tid < 128) { 
            expandAabb(mySum, sdata[tid + 128]); 
            sdata[tid] = mySum; 
        } 
        __syncthreads(); 
    }
    if (blockSize >= 128) { 
        if (tid <  64) { 
            expandAabb(mySum, sdata[tid +  64]); 
            sdata[tid] = mySum; 
        } 
        __syncthreads(); 
    }
    
    {
        // now that we are using warp-synchronous programming (below)
        // we need to declare our shared memory volatile so that the compiler
        // doesn't reorder stores to it and induce incorrect behavior.
        volatile Aabb * smem = sdata;
        if (blockSize >=  64) {
            expandAabb(mySum, &smem[tid + 32]);
            copyVola(&smem[tid], mySum);
            __syncthreads(); 
        }
        if (blockSize >=  32) { 
            expandAabb(mySum, &smem[tid + 16]);
            copyVola(&smem[tid], mySum);
            __syncthreads(); 
        }
        if (blockSize >=  16) { 
            expandAabb(mySum, &smem[tid +  8]);
            copyVola(&smem[tid], mySum);
            __syncthreads(); 
        }
        if (blockSize >=   8) { 
            expandAabb(mySum, &smem[tid +  4]);
            copyVola(&smem[tid], mySum);
            __syncthreads(); 
        }
        if (blockSize >=   4) { 
            expandAabb(mySum, &smem[tid +  2]);
            copyVola(&smem[tid], mySum);
            __syncthreads(); 
        }
        if (blockSize >=   2) { 
            expandAabb(mySum, &smem[tid +  1]);
            copyVola(&smem[tid], mySum);
            __syncthreads(); 
        }
    }
    
    // write result for this block to global mem 
    if (tid == 0) 
        g_odata[blockIdx.x] = sdata[0];
}

template <unsigned int blockSize, bool nIsPow2>
__global__ void reduceAabbByPoints_kernel(float3 *g_idata, Aabb *g_odata, unsigned int n, unsigned maxInd)
{
    extern __shared__ Aabb sdata[];

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockSize*2 + threadIdx.x;
    if(i >= maxInd) return;
    unsigned int gridSize = blockSize*2*gridDim.x;
    
    Aabb mySum; resetAabb(mySum);

    // we reduce multiple elements per thread.  The number is determined by the 
    // number of active thread blocks (via gridDim).  More blocks will result
    // in a larger gridSize and therefore fewer elements per thread
    // n <= maxInd
    while (i < n)
    {         
        expandAabb(mySum, g_idata[i]);
        // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
        if (nIsPow2 || i + blockSize < n) {
            expandAabb(mySum, g_idata[i+blockSize]); 
        }
        i += gridSize;
        if(i >= maxInd) break;
    } 

    // each thread puts its local sum into shared memory 
    sdata[tid] = mySum;
    __syncthreads();


    // do reduction in shared mem
    if (blockSize >= 512) { 
        if (tid < 256) { 
            expandAabb(mySum, sdata[tid + 256]);
            sdata[tid] = mySum; 
        } 
        __syncthreads(); 
    }
    if (blockSize >= 256) { 
        if (tid < 128) { 
            expandAabb(mySum, sdata[tid + 128]); 
            sdata[tid] = mySum; 
        } 
        __syncthreads(); 
    }
    if (blockSize >= 128) { 
        if (tid <  64) { 
            expandAabb(mySum, sdata[tid +  64]); 
            sdata[tid] = mySum; 
        } 
        __syncthreads(); 
    }
    
    {
        // now that we are using warp-synchronous programming (below)
        // we need to declare our shared memory volatile so that the compiler
        // doesn't reorder stores to it and induce incorrect behavior.
        volatile Aabb * smem = sdata;
        if (blockSize >=  64) {
            expandAabb(mySum, &smem[tid + 32]);
            copyVola(&smem[tid], mySum);
            __syncthreads(); 
        }
        if (blockSize >=  32) { 
            expandAabb(mySum, &smem[tid + 16]);
            copyVola(&smem[tid], mySum);
            __syncthreads(); 
        }
        if (blockSize >=  16) { 
            expandAabb(mySum, &smem[tid +  8]);
            copyVola(&smem[tid], mySum);
            __syncthreads(); 
        }
        if (blockSize >=   8) { 
            expandAabb(mySum, &smem[tid +  4]);
            copyVola(&smem[tid], mySum);
            __syncthreads(); 
        }
        if (blockSize >=   4) { 
            expandAabb(mySum, &smem[tid +  2]);
            copyVola(&smem[tid], mySum);
            __syncthreads(); 
        }
        if (blockSize >=   2) { 
            expandAabb(mySum, &smem[tid +  1]);
            copyVola(&smem[tid], mySum);
            __syncthreads(); 
        }
    }
    
    // write result for this block to global mem 
    if (tid == 0) 
        g_odata[blockIdx.x] = sdata[0];
}

extern "C" void bvhReduceAabbByAabb(Aabb *dst, Aabb *src, unsigned numAabbs, unsigned numBlocks, unsigned numThreads)
{
	dim3 dimBlock(numThreads, 1, 1);
    dim3 dimGrid(numBlocks, 1, 1);
	int smemSize = (numThreads <= 2) ? 2 * numThreads * sizeof(Aabb) : numThreads * sizeof(Aabb);
	
	if (isPow2(numAabbs)) {
		switch (numThreads)
		{
		case 512:
			reduceAabbByAabb_kernel<512, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, numAabbs); break;
		case 256:
			reduceAabbByAabb_kernel<256, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, numAabbs); break;
		case 128:
			reduceAabbByAabb_kernel<128, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, numAabbs); break;
		case 64:
			reduceAabbByAabb_kernel<64, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, numAabbs); break;
		case 32:
			reduceAabbByAabb_kernel<32, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, numAabbs); break;
		case 16:
			reduceAabbByAabb_kernel<16, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, numAabbs); break;
		case  8:
			reduceAabbByAabb_kernel< 8, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, numAabbs); break;
		case  4:
			reduceAabbByAabb_kernel< 4, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, numAabbs); break;
		case  2:
			reduceAabbByAabb_kernel< 2, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, numAabbs); break;
		case  1:
			reduceAabbByAabb_kernel< 1, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, numAabbs); break;
		}
	}
	else {
		switch (numThreads)
		{
		case 512:
			reduceAabbByAabb_kernel<512, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, numAabbs); break;
		case 256:
			reduceAabbByAabb_kernel<256, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, numAabbs); break;
		case 128:
			reduceAabbByAabb_kernel<128, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, numAabbs); break;
		case 64:
			reduceAabbByAabb_kernel<64, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, numAabbs); break;
		case 32:
			reduceAabbByAabb_kernel<32, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, numAabbs); break;
		case 16:
			reduceAabbByAabb_kernel<16, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, numAabbs); break;
		case  8:
			reduceAabbByAabb_kernel< 8, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, numAabbs); break;
		case  4:
			reduceAabbByAabb_kernel< 4, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, numAabbs); break;
		case  2:
			reduceAabbByAabb_kernel< 2, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, numAabbs); break;
		case  1:
			reduceAabbByAabb_kernel< 1, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, numAabbs); break;
		}
	}
}

extern "C" void bvhReduceAabbByPoints(Aabb *dst, float3 *src, unsigned numPoints, unsigned numBlocks, unsigned numThreads, unsigned maxPInd)
{
	dim3 dimBlock(numThreads, 1, 1);
    dim3 dimGrid(numBlocks, 1, 1);
	int smemSize = (numThreads < 2) ? 2 * numThreads * sizeof(Aabb) : numThreads * sizeof(Aabb);
	
	if (isPow2(numPoints)) {
		switch (numThreads)
		{
		case 512:
			reduceAabbByPoints_kernel<512, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, numPoints, maxPInd); break;
		case 256:
			reduceAabbByPoints_kernel<256, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, numPoints, maxPInd); break;
		case 128:
			reduceAabbByPoints_kernel<128, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, numPoints, maxPInd); break;
		case 64:
			reduceAabbByPoints_kernel<64, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, numPoints, maxPInd); break;
		case 32:
			reduceAabbByPoints_kernel<32, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, numPoints, maxPInd); break;
		case 16:
			reduceAabbByPoints_kernel<16, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, numPoints, maxPInd); break;
		case  8:
			reduceAabbByPoints_kernel< 8, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, numPoints, maxPInd); break;
		case  4:
			reduceAabbByPoints_kernel< 4, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, numPoints, maxPInd); break;
		case  2:
			reduceAabbByPoints_kernel< 2, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, numPoints, maxPInd); break;
		case  1:
			reduceAabbByPoints_kernel< 1, true><<< dimGrid, dimBlock, smemSize >>>(src, dst, numPoints, maxPInd); break;
		}
	}
	else {
		switch (numThreads)
		{
		case 512:
			reduceAabbByPoints_kernel<512, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, numPoints, maxPInd); break;
		case 256:
			reduceAabbByPoints_kernel<256, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, numPoints, maxPInd); break;
		case 128:
			reduceAabbByPoints_kernel<128, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, numPoints, maxPInd); break;
		case 64:
			reduceAabbByPoints_kernel<64, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, numPoints, maxPInd); break;
		case 32:
			reduceAabbByPoints_kernel<32, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, numPoints, maxPInd); break;
		case 16:
			reduceAabbByPoints_kernel<16, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, numPoints, maxPInd); break;
		case  8:
			reduceAabbByPoints_kernel< 8, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, numPoints, maxPInd); break;
		case  4:
			reduceAabbByPoints_kernel< 4, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, numPoints, maxPInd); break;
		case  2:
			reduceAabbByPoints_kernel< 2, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, numPoints, maxPInd); break;
		case  1:
			reduceAabbByPoints_kernel< 1, false><<< dimGrid, dimBlock, smemSize >>>(src, dst, numPoints, maxPInd); break;
		}
	}
}