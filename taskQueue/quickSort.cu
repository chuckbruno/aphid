#include "hip/hip_runtime.h"
/*
 *  sort node count and range
 *
 *  1    0,13
 *  2    0,6  7,13
 *  4    0,3  4,6  7,10  11,13
 *  8    0,1  2,3  4,5   5,6   7,8  9,10  11,12  13,13
 *
 *  each node consumes one spawns two or one or zero
 *  level n needs max 2^n nodes
 *  each level has node count and range updated by atomic
 *  
 *  taskIn  1  0,13 |
 *  taskOut 3  0,13 | 0,6  7,13 
 *
 *  taskIn  3  0,13   0,6  7,13 |
 *  taskOut 7  0,13   0,6  7,13 | 0,3   4,6  7,10  11,13
 *
 *  taskIn  7  0,13   0,6  7,13   0,3   4,6  7,10  11,13 |
 *  taskOut 15 0,13   0,6  7,13   0,3   4,6  7,10  11,13 | 0,1  2,3  4,5   5,6   7,8  9,10  11,12  13,13
 *
 *  when taskIn.tbid > taskOut.qtail means no more work is available
 *  no more work is available doesn't mean the job is done
 *  still could be more task added, need workDoneCounter
 *  once workDoneCounter > taskOut.qtail, there is nothing left to do
 *  intially task out is a copy of task in
 *  new tasks will be added to end
 *  each time a task is done first enqueue() of needed, 
 *  then check if work is done, if not then dequeue()
 *  
 */
 
#include "quickSort.cuh"
#include "OddEvenSort.cuh"
#include "bvh_common.h"
#include "CudaBase.h"

extern "C" {
void cu_testQuickSort(void * q,
                    uint * idata,
                    uint * nodes, 
                    int * elements,
                    SimpleQueueInterface * qi,
                    uint numElements,
                    uint * workBlocks,
                    uint * loopbuf,
                    int * headtailperloop)
{
    //hipDeviceSynchronize();
    
    simpleQueue::SimpleQueue * queue = (simpleQueue::SimpleQueue *)q;
    simpleQueue::init_kernel<<< 1,32 >>>(queue, 1, elements);
    
    const int tpb = 256;
    dim3 block(tpb, 1, 1);
    const unsigned nblk = 1024;
    dim3 grid(nblk, 1, 1);
    
    oddEvenSort::OddEvenSortTask oes;
    oddEvenSort::DataInterface oesd;
    oesd.idata = idata;
    oesd.nodes = (int2 *)nodes;
    
    int lpb = 1 + numElements>>10;
    
    quickSort_test_kernel<simpleQueue::SimpleQueue, oddEvenSort::OddEvenSortTask, oddEvenSort::DataInterface, 24><<<grid, block, 16320>>>(queue,
                                oes,
                                oesd,
                                qi,
                                workBlocks,
                                loopbuf,
                                (int4 *)headtailperloop,
                                lpb,
                                255);
    
    CudaBase::CheckCudaError("q sort");
}

}

