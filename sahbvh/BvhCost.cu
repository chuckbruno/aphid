#include "hip/hip_runtime.h"
#include "BvhCost.cuh"
#include <iostream>
namespace bvhcost {

void computeTraverseCost(float * costs,
        int2 * nodes,
        int * nodeNumPrimitives,
	    Aabb * nodeAabbs,
        uint n)
{
    const int tpb = 256;
    dim3 block(tpb, 1, 1);
    unsigned nblk = iDivUp(n, tpb);
    dim3 grid(nblk, 1, 1);
    
    computeTraverseCost_kernel<<< grid, block>>>(costs,
                            nodes,
                            nodeNumPrimitives,
                            nodeAabbs,
                            n);
}

void countPrimitviesInNodeAtLevel(int * nodeNumPrimitives,
        int * nodeLevels,
        int2 * nodes,
        int level,
	    uint n)
{
    const int tpb = 512;
    dim3 block(tpb, 1, 1);
    unsigned nblk = iDivUp(n, tpb);
    dim3 grid(nblk, 1, 1);
    
    countPrimitviesInNodeAtLevel_kernel<<< grid, block>>>(nodeNumPrimitives,
        nodeLevels,
        nodes,
        level,
	    n);
}

}
