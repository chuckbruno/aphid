#include "hip/hip_runtime.h"
#include "CubeRender.cuh"

namespace cuber {

void setRenderRect(int * src)
{ hipMemcpyToSymbol(HIP_SYMBOL(c_renderRect), src, 16); }

void setFrustum(float * src)
{ hipMemcpyToSymbol(HIP_SYMBOL(c_frustumVec), src, 72); }

void render(uint * pix,
            float * depth,
            int blockx,
            int gridx, int gridy)
{
    dim3 block(blockx, blockx, 1);
    dim3 grid(gridx, gridy, 1);
    
    oneCube_kernel<<< grid, block >>>(pix, 
        depth);
}

void drawPyramid(uint * color,
                float * depth,
                int blockx,
                int gridx, int gridy,
				void * planes,
				void * bounding)
{
    dim3 block(blockx, blockx, 1);
    dim3 grid(gridx, gridy, 1);
    
    onePyrmaid_kernel<<< grid, block >>>(color, 
        depth,
        (float4 *)planes,
        (Aabb *)bounding);
}

void drawVoxel(uint * color,
                float * depth,
                int blockx,
                int gridx, int gridy,
				void * voxels)
{
    dim3 block(blockx, blockx, 1);
    dim3 grid(gridx, gridy, 1);
    
    oneVoxel_kernel<<< grid, block, 14096 >>>(color, 
        depth,
        (Voxel *)voxels);
}

const float cubefaces[] = {
-1, 0, 0,
 1, 0, 0,
 0,-1, 0,
 0, 1, 0,
 0, 0,-1,
 0, 0, 1
};

void setBoxFaces()
{ hipMemcpyToSymbol(HIP_SYMBOL(c_ray_box_face), cubefaces, 72); }

}
