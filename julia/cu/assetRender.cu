#include "assetRender.cuh"

namespace assr {

void setRenderRect(int * src)
{ hipMemcpyToSymbol(HIP_SYMBOL(c_renderRect), src, 16); }

void setFrustum(float * src)
{ hipMemcpyToSymbol(HIP_SYMBOL(c_frustumVec), src, 72); }

void drawCube(uint * color,
                float * nearDepth,
                float * farDepth,
				int blockx,
                int gridx, int gridy,
                void * branches,
				void * leaves,
				void * ropes,
				int * indirections,
				void * primitives
                )
{
    dim3 block(blockx, blockx, 1);
    dim3 grid(gridx, gridy, 1);
    
    if(blockx == 8) {
        assetBox_kernel<64> <<< grid, block, 16000 >>>(color, 
            nearDepth,
            farDepth,
            (NTreeBranch4 *)branches,
            (NTreeLeaf *)leaves,
            (Rope *)ropes,
            indirections,
            (Voxel *)primitives);
	}
	else if(blockx == 16) {
	    assetBox_kernel<256> <<< grid, block, 16000 >>>(color, 
            nearDepth,
            farDepth,
            (NTreeBranch4 *)branches,
            (NTreeLeaf *)leaves,
            (Rope *)ropes,
            indirections,
            (Voxel *)primitives);
	}
}

}
