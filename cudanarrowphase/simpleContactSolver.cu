#include "hip/hip_runtime.h"
/*
 * reference
 * http://www.cs.cornell.edu/courses/cs5643/2013sp/a3Rigidbody/
 * http://www-cs-students.stanford.edu/~eparker/files/PhysicsEngine/
 * http://www.rowlhouse.co.uk/jiggle/index.html
 * http://www.richardtonge.com/papers/Tonge-2012-MassSplittingForJitterFreeParallelRigidBodySimulation-preprint.pdf
 */

#include "contactSolverCommon.cuh"
#include <CudaBase.h>

#define DEFORMABILITY 0.0134f
#define ENABLE_DEFORMABILITY 0
#define VERYLARGE_INT 16777215 // 1<<24 - 1
#define VERYLARGE_INT_M1 16777214 
#define VERYVERYLARGE_INT 1073741823 // 1<<30 - 1

__constant__ float CSpeedLimit;

inline __device__ void computeBodyAngularVelocity(float3 & angularVel,
                                                  float3 averageLinearVel,
                                                  float3 * position,
                                                  float3 * velocity,
                                                  uint4 ind)
{
    float3 center;
	float3_average4(center, position, ind);
	
	float3 omega[4];
// omega = r cross v
// v = omega cross r
    omega[0] = float3_cross(float3_difference(position[ind.x], center), float3_difference(velocity[ind.x], averageLinearVel));
    omega[1] = float3_cross(float3_difference(position[ind.y], center), float3_difference(velocity[ind.y], averageLinearVel));
    omega[2] = float3_cross(float3_difference(position[ind.z], center), float3_difference(velocity[ind.z], averageLinearVel));
    omega[3] = float3_cross(float3_difference(position[ind.w], center), float3_difference(velocity[ind.w], averageLinearVel));
    
	float3_average4_direct(angularVel, omega);
}

inline __device__ void computeBodyVelocities1(uint * pointStarts, 
                                                uint * indexStarts, 
                                                uint4 * indices, 
                                                uint ind,
                                                float3 * position,
                                                float3 * velocity, 
                                                float3 & linearVelocity, 
                                                float3 & angularVelocity)
{
    const uint4 ia = computePointIndex(pointStarts, indexStarts, indices, ind);
	
	float3_average4(linearVelocity, velocity, ia);
	computeBodyAngularVelocity(angularVelocity, linearVelocity, position, velocity, ia);
}

inline __device__ void computeBodyCenter(float3 & center, 
                                uint iBody,
                                float3 * position,
                                uint4 * indices,
                                uint * pointStarts,
                                uint * indexStarts)
{
    const uint4 iVert = computePointIndex(pointStarts, indexStarts, indices, iBody);
	float3_average4(center, position, iVert);
}

inline __device__ uint getBodyCountAt(uint ind, uint * count)
{
    uint cur = ind;
    for(;;) {
        if(count[cur] > 0) return count[cur];
        cur--;
    }
}

inline __device__ float computeRelativeVelocity(float3 nA,
                            float3 nB,
                            float3 linearVelocityA, 
                            float3 linearVelocityB,
                            float3 torqueA,
                            float3 torqueB,
                            float3 angularVelocityA, 
                            float3 angularVelocityB)
{
    return float3_dot(linearVelocityA, nA) +
            float3_dot(linearVelocityB, nB);// +
            // float3_dot(torqueA, angularVelocityA) +
            // float3_dot(torqueB, angularVelocityB);
}

inline __device__ void deformMotion(float3 & dst,
                                    float3 r, 
                                    float3 n,
                                    float3 omega)
{
// v = omega X r 
    dst = float3_cross(omega, r);
    float l = float3_length2(dst);
    float lr = float3_length(r);
// limit size of rotation
    if(l> lr * .59f) l = lr * .59f;
    if(l>1e-2) dst = float3_normalize(dst);
    dst = scale_float3_by(dst, l);
    dst = float3_add(dst, scale_float3_by(n, lr));
    dst = scale_float3_by(dst, DEFORMABILITY);
}

inline __device__ void addDeltaVelocity(float3 & dst, 
        float3 deltaLinearVelocity,
        float3 deltaAngularVelocity,
        float3 normal, 
        float3 r,
        BarycentricCoordinate * coord)
{
    dst = float3_add(dst, deltaLinearVelocity);
#if ENABLE_DEFORMABILITY
    float3 vRot;
    deformMotion(vRot, r, normal, deltaAngularVelocity);
    
// distribure by weight to vex, then sum by weight from vex
    float wei = coord->x * coord->x;
    wei = wei > 1.f ? 1.f : wei;
    dst = float3_add(dst, scale_float3_by(vRot, wei));
    
    wei = coord->y * coord->y;
    wei = wei > 1.f ? 1.f : wei;
    dst = float3_add(dst, scale_float3_by(vRot, wei));
    
    wei = coord->z * coord->z;
    wei = wei > 1.f ? 1.f : wei;
    dst = float3_add(dst, scale_float3_by(vRot, wei));
    
    wei = coord->w * coord->w;
    wei = wei > 1.f ? 1.f : wei;
    dst = float3_add(dst, scale_float3_by(vRot, wei));
#endif
}

inline __device__ float getPntTetWeight(uint pnt, 
                            uint4 tet, 
                            BarycentricCoordinate coord)
{
    if(pnt == tet.x) return coord.x;
    if(pnt == tet.y) return coord.y;
    if(pnt == tet.z) return coord.z;
    return coord.w;
}

__global__ void writeContactIndex_kernel(KeyValuePair * dstInd, 
                                    uint * srcInd, 
                                    uint n, uint bufferLength)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= bufferLength) return;
	
	if(ind < n) {
	    dstInd[ind].key = srcInd[ind];
	    dstInd[ind].value = ind >> 1;
	}
	else {
	    dstInd[ind].key = VERYVERYLARGE_INT;
	    dstInd[ind].value = VERYVERYLARGE_INT;
	}
}

__global__ void computeSplitBufLoc_kernel(uint2 * splits, 
                                    uint2 * srcPairs, 
                                    KeyValuePair * bodyPairHash, 
                                    uint bufLength)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= bufLength) return;
	
	const uint iPair = bodyPairHash[ind].value;
	if(srcPairs[iPair].x == bodyPairHash[ind].key) {
	    splits[iPair].x = ind;
	}
	else {
	    splits[iPair].y = ind;
	}
}

__global__ void countBody_kernel(uint * dstCount,
                                    KeyValuePair * srcInd, 
                                    uint maxInd)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= maxInd) return;
	
	dstCount[ind] = 0;
	
	const uint a = srcInd[ind].key;
	
	int isFirst = 0;
	
	if(ind < 1) isFirst = 1;
	else if(srcInd[ind - 1].key != a) isFirst = 1;
	
	if(!isFirst) return;
	
	dstCount[ind] = 1;

	unsigned cur = ind;
// check backward
	for(;;) {
	    if(cur == maxInd - 1) return;
	    cur++;
	    if(srcInd[cur].key != a) return;
	    dstCount[ind]++;
	}	
}

__global__ void computeSplitInvMass_kernel(float * invMass,
                                        uint2 * splits,
                                        uint2 * pairs,
                                        float * mass,
	                                    uint4 * indices,
	                                    uint * pointStart,
	                                    uint * indexStart,
	                                    uint * bodyCount,
	                                    uint4 * tet,
                                        uint maxInd)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= maxInd) return;
	
	const uint iPair = ind>>1;
	const int isRgt = (ind & 1);
	
	uint4 ia;
	uint dstInd;
	if(isRgt > 0) {
	    dstInd = splits[iPair].y;
	    ia = computePointIndex(pointStart, indexStart, indices, pairs[iPair].y);
	}
	else {
	    dstInd = splits[iPair].x;
	    ia = computePointIndex(pointStart, indexStart, indices, pairs[iPair].x);
	}
	
	tet[ind] = ia;
	
	uint n = getBodyCountAt(dstInd, bodyCount);
	
	invMass[dstInd] = (float)n / (mass[ia.x] + mass[ia.y] + mass[ia.z] + mass[ia.w]);
}

__global__ void clearDeltaVelocity_kernel(float3 * deltaLinVel, 
                                        float3 * deltaAngVel, 
                                        uint maxInd)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= maxInd) return;
	
	deltaLinVel[ind] = make_float3(0.f, 0.f, 0.f);
	// deltaAngVel[ind] = make_float3(0.f, 0.f, 0.f);
}

__global__ void solveContact_kernel(ContactConstraint* constraints,
                        float3 * deltaLinearVelocity,
	                    float3 * deltaAngularVelocity,
	                    uint2 * pairs,
                        uint2 * splits,
	                    float * splitMass,
	                    ContactData * contacts,
	                    float3 * positions,
                        float3 * velocities,
                        uint4 * indices,
                        uint * pointStarts,
                        uint * indexStarts,
                        uint maxInd,
                        float * deltaJ,
                        int maxNIt,
                        int it)
{
    __shared__ float3 sVel[SOLVECONTACT_TPB];
    __shared__ float3 sN[SOLVECONTACT_TPB];
    
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= maxInd) return;
	
	const uint iContact = ind>>1;
	
	uint splitInd = splits[iContact].x;
	uint iBody = pairs[iContact].x;
	BarycentricCoordinate coord = constraints[iContact].coordA;
	
	if((threadIdx.x & 1)>0) {
	    splitInd = splits[iContact].y;
	    iBody = pairs[iContact].y;
	    coord = constraints[iContact].coordB;
	}

// initial velocities
    uint4 ia = computePointIndex(pointStarts, indexStarts, indices, iBody);
    
    float3 velA;
    interpolate_float3i(velA, ia, velocities, &coord);
    
	float3 nA = constraints[iContact].normal;
	float3 rA;// = contacts[iContact].localA;
	
	if((ind & 1)>0) {
	    nA = float3_reverse(nA);
	    // rA = contacts[iContact].localB;
	}
	
// N pointing inside object
// T = r X N	
	float3 torqueA = float3_cross(rA, nA);
	
	addDeltaVelocity(velA, 
        deltaLinearVelocity[splitInd],
        deltaAngularVelocity[splitInd],
        nA, rA,
        &coord);
    
    sN[threadIdx.x] = nA;
    sVel[threadIdx.x] = velA;
    __syncthreads();
    
    uint iLeft = (threadIdx.x>>1)<<1;
    uint iRight = iLeft + 1;
	
	float J = computeRelativeVelocity1(sN[iLeft], sN[iRight],
	                        sVel[iLeft], sVel[iRight]);

	J += constraints[iContact].relVel;
	J *= constraints[iContact].Minv;
	
	float prevSum = constraints[iContact].lambda;
	float updated = prevSum;
	updated += J;
	if(updated < 0.f) updated = 0.f;
	
	if((threadIdx.x & 1)==0) constraints[iContact].lambda = updated;
	
	J = updated - prevSum;
	
	if((threadIdx.x & 1)==0) deltaJ[iContact * maxNIt + it] = J;
	
	const float invMassA = splitMass[splitInd];
	
	//applyImpulse(deltaLinearVelocity[splitInd], J * invMassA, nA);
	//applyImpulse(deltaAngularVelocity[splitInd], J * invMassA, torqueA);
}

__global__ void averageVelocities_kernel(float3 * linearVelocity,
                        float3 * angularVelocity,
                        uint * bodyCount, 
                        KeyValuePair * srcInd,
                        uint maxInd)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= maxInd) return;
	
	uint c = bodyCount[ind];
	if(c < 1) return;
	
	uint a = srcInd[ind].key;
	
	float3 linSum = linearVelocity[ind];
	// float3 angSum = angularVelocity[ind];

	unsigned cur = ind;
// add up backward
	for(;;) {
	    if(cur == maxInd - 1) break;
	    cur++;
	    if(srcInd[cur].key != a) break;
	    
	    linSum = float3_add(linSum, linearVelocity[cur]);
	    // angSum = float3_add(angSum, angularVelocity[cur]);
	}

	if(c > 1) {
	    linSum = scale_float3_by(linSum, 1.f / (float)c);
	    // angSum = scale_float3_by(angSum, 1.f / (float)c);
	}
	
	linearVelocity[ind] = linSum;
	// angularVelocity[ind] = angSum;
	
	cur = ind;
// write backward
	for(;;) {
	    if(cur == maxInd - 1) break;
	    cur++;
	    if(srcInd[cur].key != a) break;
	    
	    linearVelocity[cur] = linSum;
	    // angularVelocity[cur] = angSum;
	}
}

__global__ void resetPointTetHash_kernel(KeyValuePair * pntTetHash,
	                uint maxInd)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= maxInd) return;
    pntTetHash[ind].key = VERYLARGE_INT;
}

__global__ void writePointTetHash_kernel(KeyValuePair * pntTetHash,
	                uint2 * pairs,
	                uint2 * splits,
	                uint * bodyCount,
	                uint4 * tet,
                    uint maxInd)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= maxInd) return;
	
	const uint istart = ind * 4;
	
	const unsigned iContact = ind>>1;
	
	uint splitInd = splits[iContact].x;
	uint iBody = pairs[iContact].x;
	
	if(ind & 1) {
	    splitInd = splits[iContact].y;
	    iBody = pairs[iContact].y;
	}
    
	KeyValuePair kv;  
    
	if(bodyCount[splitInd] < 1) {
// redundant
        pntTetHash[istart].key = VERYLARGE_INT;
        pntTetHash[istart + 1].key = VERYLARGE_INT;
        pntTetHash[istart + 2].key = VERYLARGE_INT;
        pntTetHash[istart + 3].key = VERYLARGE_INT;
	}
	else {
	    uint4 ia = tet[ind];
        kv.value = ind;
	    kv.key = ia.x;
        pntTetHash[istart  ] = kv;
        
        kv.key = ia.y;
	    pntTetHash[istart+1] = kv;
        
        kv.key = ia.z;
	    pntTetHash[istart+2] = kv;
	    
        kv.key = ia.w;
	    pntTetHash[istart+3] = kv;
	}
}

__global__ void updateVelocity_kernel(float3 * dstVelocity,
                    float3 * deltaLinearVelocity,
	                float3 * deltaAngularVelocity,
	                KeyValuePair * pntTetHash,
                    uint2 * pairs,
                    uint2 * splits,
                    ContactConstraint * constraints,
                    ContactData * contacts,
                    float3 * position,
                    uint4 * indices,
                    uint * objectPointStarts,
                    uint * objectIndexStarts,
                    uint maxInd)
{
    unsigned ind = blockIdx.x*blockDim.x + threadIdx.x;
	if(ind >= maxInd) return;
	
	const uint iPnt = pntTetHash[ind].key;
	
	if(iPnt > VERYLARGE_INT_M1) return;
	
	int isFirst = 0;
	
	if(ind < 1) isFirst = 1;
	else if(pntTetHash[ind - 1].key != iPnt) isFirst = 1;
	
	if(!isFirst) return;
	
	float3 sumLinVel = make_float3(0.f, 0.f, 0.f);
	float count = 0.f;
	uint cur = ind;
	uint iContact, splitInd;
	
	float3 normal;
#if ENABLE_DEFORMABILITY
    uint iBody;
    float3 r, vRot;
    float weight;
    uint4 iTet;
	BarycentricCoordinate coord;
#endif
	for(;;) {
	    iContact = pntTetHash[cur].value>>1;
	
        if(pntTetHash[cur].value & 1) {
            splitInd = splits[iContact].y;
            
#if ENABLE_DEFORMABILITY
            iBody = pairs[iContact].y;
            coord = constraints[iContact].coordB;
            r = contacts[iContact].localB;
            normal = float3_reverse(constraints[iContact].normal);
#endif
            
        }
        else {
            splitInd = splits[iContact].x;
	    
#if ENABLE_DEFORMABILITY
            iBody = pairs[iContact].x;
            coord = constraints[iContact].coordA;
            r = contacts[iContact].localA;
            normal = constraints[iContact].normal;
#endif
        }
        
        sumLinVel = float3_add(sumLinVel, deltaLinearVelocity[splitInd]);
             
#if ENABLE_DEFORMABILITY
        iTet = computePointIndex(objectPointStarts, objectIndexStarts, indices, iBody);
        weight = getPntTetWeight(iPnt, iTet, coord);

        deformMotion(vRot, r, normal, deltaAngularVelocity[splitInd]);
// weighted by vex coord        
        vRot = scale_float3_by(vRot, weight);

        sumLinVel = float3_add(sumLinVel, vRot);
#endif
        count += 1.f;
        
        if(cur == maxInd - 1) break;
	    cur++;
	    if(pntTetHash[cur].key != iPnt) break;
	}
	
	if(count > 1.f)
	    sumLinVel = scale_float3_by(sumLinVel, 1.f / count);

    //float3 a = dstVelocity[iPnt];
	//float3_add_inplace(a, sumLinVel);
    float speed = float3_length(sumLinVel);
// limit speed here
    if(speed > CSpeedLimit) float3_scale_inplace(sumLinVel, CSpeedLimit / speed);
    //dstVelocity[iPnt] = a;
    float3_add_inplace(dstVelocity[iPnt], sumLinVel);
}

extern "C" {
    
void simpleContactSolverWriteContactIndex(KeyValuePair * dstInd, 
                                    uint * srcInd, 
                                    uint n, uint bufferLength)
{
    dim3 block(512, 1, 1);
    unsigned nblk = iDivUp(bufferLength, 512);
    dim3 grid(nblk, 1, 1);
    
    writeContactIndex_kernel<<< grid, block >>>(dstInd, 
                                                srcInd,
                                                n, bufferLength);
}

void simpleContactSolverComputeSplitBufLoc(uint2 * splits, 
                                    uint2 * srcPairs, 
                                    KeyValuePair * bodyPairHash, 
                                    uint bufLength)
{
    dim3 block(512, 1, 1);
    unsigned nblk = iDivUp(bufLength, 512);
    dim3 grid(nblk, 1, 1);
    
    computeSplitBufLoc_kernel<<< grid, block >>>(splits, 
                                        srcPairs, 
                                        bodyPairHash, 
                                        bufLength);
}

void simpleContactSolverCountBody(uint * dstCount,
                                    KeyValuePair * srcInd, 
                                    uint num)
{
    dim3 block(512, 1, 1);
    unsigned nblk = iDivUp(num, 512);
    dim3 grid(nblk, 1, 1);
    
    countBody_kernel<<< grid, block >>>(dstCount,
                                     srcInd, 
                                       num);
}

void simpleContactSolverComputeSplitInverseMass(float * invMass,
                                        uint2 * splits,
                                        uint2 * pairs,
                                        float * mass,
	                                    uint4 * ind,
	                                    uint * perObjPointStart,
	                                    uint * perObjectIndexStart,
                                        uint * bodyCount, 
                                        uint4 * tet,
                                        uint bufLength)
{
    dim3 block(512, 1, 1);
    unsigned nblk = iDivUp(bufLength, 512);
    dim3 grid(nblk, 1, 1);
    
    computeSplitInvMass_kernel<<< grid, block >>>(invMass,
                                        splits,
                                        pairs,
                                        mass,
	                                    ind,
	                                    perObjPointStart,
	                                    perObjectIndexStart,
	                                    bodyCount, 
	                                    tet,
	                                    bufLength);
}

void simpleContactSolverClearDeltaVelocity(float3 * deltaLinVel, 
                                        float3 * deltaAngVel, 
                                        uint bufLength)
{
    dim3 block(512, 1, 1);
    unsigned nblk = iDivUp(bufLength, 512);
    dim3 grid(nblk, 1, 1);
    
    clearDeltaVelocity_kernel<<< grid, block >>>(deltaLinVel,
                                     deltaAngVel, 
                                       bufLength);
}

void simpleContactSolverAverageVelocities(float3 * linearVelocity,
                        float3 * angularVelocity,
                        uint * bodyCount, 
                        KeyValuePair * srcInd,
                        uint numBodies)
{
    dim3 block(512, 1, 1);
    unsigned nblk = iDivUp(numBodies, 512);
    dim3 grid(nblk, 1, 1);
    
    averageVelocities_kernel<<< grid, block >>>(linearVelocity,
                        angularVelocity,
                        bodyCount, 
                        srcInd,
                        numBodies);
}

void simpleContactSolverWritePointTetHash(KeyValuePair * pntTetHash,
	                uint2 * pairs,
	                uint2 * splits,
	                uint * bodyCount,
	                uint4 * tet,
	                uint numBodies,
	                uint bufLength)
{
    dim3 block(512, 1, 1);
    unsigned nblk = iDivUp(bufLength, 512);
    dim3 grid(nblk, 1, 1);
    
    resetPointTetHash_kernel<<< grid, block >>>(pntTetHash,
                                                bufLength);
    grid.x = iDivUp(numBodies, 512);
    writePointTetHash_kernel<<< grid, block >>>(pntTetHash,
	                pairs,
	                splits,
	                bodyCount,
	                tet,
	                numBodies);
}

}

namespace contactsolver {
	void setSpeedLimit(float x)
	{ hipMemcpyToSymbol(HIP_SYMBOL(CSpeedLimit), &x, 4); }
    
    void updateImpulse(float3 * dstImpulse,
                    float3 * deltaLinearVelocity,
	                float3 * deltaAngularVelocity,
	                KeyValuePair * pntTetHash,
                    uint2 * pairs,
                    uint2 * splits,
                    ContactConstraint * constraints,
                    ContactData * contacts,
                    float3 * position,
                    uint4 * indices,
                    uint * objectPointStarts,
                    uint * objectIndexStarts,
                    uint numPoints)
{
    uint tpb = 256;

    dim3 block(tpb, 1, 1);
    unsigned nblk = iDivUp(numPoints, tpb);
    dim3 grid(nblk, 1, 1);
    
    updateVelocity_kernel<<< grid, block >>>(dstImpulse,
                    deltaLinearVelocity,
	                deltaAngularVelocity,
	                pntTetHash,
                    pairs,
                    splits,
                    constraints,
                    contacts,
                    position,
                    indices,
                    objectPointStarts,
                    objectIndexStarts,
                    numPoints);
}
}


//:~
